
#include <hip/hip_runtime.h>
#define IDX3(X, n1, n2, n3, i1, i2, i3) (X[(i1)*((n2)*(n3)) + (i2)*(n3) + (i3)])

template<class T>
__device__ void im2col_ker(const T *im, T *patches,
                           int im_ni, int im_nj, int nimgs,
                           int p_ni, int p_nj, int npatches)
{
	int total_threads = gridDim.x * blockDim.x;
    int patch = blockIdx.x * blockDim.x + threadIdx.x;
    int patches_per_img = npatches / nimgs;

	for (; patch < npatches; patch += total_threads) {
		int im_k = patch / patches_per_img;  /* image index */
		int im_j0 = patch / (im_ni - p_ni + 1);  /* patch topleft j in image */
		int im_i0 = patch % (im_ni - p_ni + 1);  /* patch topleft i in image */

		for (int pj = 0; pj < p_nj; ++pj) {
			for (int pi = 0; pi < p_ni; ++pi) {
				IDX3(patches, npatches, p_nj, p_ni,
							  patch, pj, pi)
						= IDX3(im, nimgs, im_nj, im_ni,
								   im_k, im_j0 + pj, im_i0 + pi);
			}
		}
	}
}


template<class T>
__device__ void scol2im_ker(T *im, const T *patches,
                            int im_ni, int im_nj, int nimgs,
                            int p_ni, int p_nj, int npatches)
{
	int total_threads = gridDim.x * blockDim.x;
    int pixel = blockIdx.x * blockDim.x + threadIdx.x;
	int valid_nj = im_nj - p_nj + 1;
	int valid_ni = im_ni - p_ni + 1;
    int npixels = nimgs * im_nj * im_ni;
    int patches_per_img = npatches / nimgs;

	for (; pixel < npixels; pixel += total_threads) {
		T x = 0;

		int im_k = pixel / (im_ni * im_nj);  /* image index */
		int im_j = pixel / im_ni;  /* pixel in image */
		int im_i = pixel % im_ni;

		for (int pj = 0; pj < p_nj; ++pj) {
			for (int pi = 0; pi < p_ni; ++pi) {
				int im_pj = im_j - pj;  /* topleft of patch in image */
				int im_pi = im_i - pi;  /* topleft of patch in image */
				if (im_pi < 0 || im_pj < 0 || 
					im_pj >= valid_nj || im_pi >= valid_ni)
						continue;

				int patch = im_k * patches_per_img + im_pj * valid_ni + im_pi;
				x += IDX3(patches, npatches, p_nj, p_ni,
								   patch, pj, pi);
			}
		}

		IDX3(im, nimgs, im_nj, im_ni,
				 im_k, im_j, im_i) = x;
	}
}

__global__ void im2col_d(const double *im, double *patches,
                         int im_ni, int im_nj, int nimgs,
                         int p_ni, int p_nj, int npatches)
{
    im2col_ker<double>(im, patches,
                       im_ni, im_nj, nimgs,
                       p_ni, p_nj, npatches);
}

__global__ void scol2im_d(double *im, const double *patches,
                          int im_ni, int im_nj, int nimgs,
                          int p_ni, int p_nj, int npatches)
{
    scol2im_ker<double>(im, patches,
                        im_ni, im_nj, nimgs,
                        p_ni, p_nj, npatches);
}

__global__ void im2col_f(const float *im, float *patches,
                         int im_ni, int im_nj, int nimgs,
                         int p_ni, int p_nj, int npatches)
{
    im2col_ker<float>(im, patches,
                      im_ni, im_nj, nimgs,
                      p_ni, p_nj, npatches);
}

__global__ void scol2im_f(float *im, const float *patches,
                          int im_ni, int im_nj, int nimgs,
                          int p_ni, int p_nj, int npatches)
{
    scol2im_ker<float>(im, patches,
                       im_ni, im_nj, nimgs,
                       p_ni, p_nj, npatches);
}

