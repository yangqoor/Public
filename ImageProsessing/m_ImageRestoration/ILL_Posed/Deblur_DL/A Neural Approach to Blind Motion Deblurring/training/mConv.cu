#include "hip/hip_runtime.h"
/* 
   Very little error checking. Use with Caution.

   --Ayan Chakrabarti <ayanc@ttic.edu>
*/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <stdint.h>

#define F float

#define NUMT 1024
#define tid threadIdx.x



void __global__ mConv(F * y, F * x, F * k,
		      int ysz, int ksz, int bsz) {


	extern __shared__ F myk[];
	F sum, *kp;
	int bid, n, i, j, xsz, n2; 

	xsz = ysz+ksz-1;

	/* Responsible for finding y(n,bid) = x(:,bid)*k(:,bid) */
	bid = blockIdx.x;
	n = bid % bsz; bid /= bsz;
	n = n * NUMT + tid;


	/* Make a copy of k(:,bid) into shared memory */
	for(i = 0; i < ksz*ksz; i+= NUMT)
		if(i + tid < ksz*ksz)
			myk[i+tid] = k[bid*ksz*ksz+i+tid];
	
	__syncthreads();

	if(n < ysz*ysz) {
		sum = 0; 
		kp = &myk[ksz*ksz-1];
		n2 = n + (n/ysz)*(xsz-ysz); 
		for(j = 0; j < ksz; j++)
			for(i = 0; i < ksz; i++)
				sum += (*(kp--)) *
					x[ bid*xsz*xsz +
					   j*xsz + i + n2 ];
		y[bid*ysz*ysz+n] = sum;
	}
}


F * getGPUmem(const char * name) {

	const mxGPUArray * tmp;
	F * dptr;

	if(!mxIsGPUArray(mexGetVariablePtr("caller",name)))
		mexPrintf("%s is not on gpu!\n",name);

	tmp = mxGPUCreateFromMxArray(mexGetVariablePtr("caller",name));
	dptr = (F*) mxGPUGetDataReadOnly(tmp);
	mxGPUDestroyGPUArray(tmp);

	return (F*) dptr;
}

/* function mConv */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

	F * y, * x, * k;
	int ysz, ksz, bsz, nsamp, numB, sharesz;

	ysz = (int) mxGetScalar(mexGetVariablePtr("caller","ysz"));
	ksz = (int) mxGetScalar(mexGetVariablePtr("caller","ksz"));
	nsamp = mxGetScalar(mexGetVariablePtr("caller","nsamp"));

	bsz = (ysz*ysz+NUMT-1)/NUMT; numB = bsz*nsamp; sharesz = ksz*ksz*sizeof(F);

	y = getGPUmem("y"); x = getGPUmem("x"); k = getGPUmem("k");

	mConv<<<numB,NUMT,sharesz>>>(y,x,k,ysz,ksz,bsz);
}
